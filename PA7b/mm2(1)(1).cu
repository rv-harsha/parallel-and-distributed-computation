#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define HEIGHT 1024
#define WIDTH 1024
#define BLOCK_SIZE 32


__global__ void matrix_mult(int *a, int *b, int *c){
	int threadRow = threadIdx.y;
	int threadCol = threadIdx.x;

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = /*your code here*/;

	int c_val = 0;
	for (int i = 0; i<(WIDTH/BLOCK_SIZE); i++) {
		__shared__ int a_share[BLOCK_SIZE][BLOCK_SIZE];
		/*your code here to declare b_share*/

		// each thread reads one element from both A and B matrices into the shared sub-matrices
		a_share[threadRow][threadCol] = a[/*your code here*/];
		b_share[threadRow][threadCol] = b[/*your code here*/];

		// make sure the sub-matrices are loaded before starting the computation
		__syncthreads();

		for (int i=0; i<BLOCK_SIZE; i++) {
			c_val += a_share[/*your code here*/][i] * b_share[i][/*your code here*/];
		}

		// make sure every thread is done computing before loading new sub-matrices
		__syncthreads();

	}

	c[row * WIDTH + col] = c_val;
}

int main(){
  int i;
  int *a = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
	int *b = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
  int *c = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
	for(i=0; i<WIDTH * HEIGHT; i++){
		a[i]=1;
		b[i]=2;
  	}

	int *gpu_a, *gpu_b, *gpu_c;
	/*your code here to malloc gpu_a, gpu_b, gpu_c on device*/

	struct timespec start, stop;
	double time;

	/*your code here to copy a and b from host to device*/

	/*your code here to create dimGrid and dimBlock*/

	if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}

	matrix_mult<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
	/*your code here to copy gpu_c from device to host*/

	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}
	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("time is %f ns\n", time*1e9);

	printf("c[451][451]=%d\n", c[451*1024+451]);

	free(a);
	free(b);
	free(c);
	/*your code here to free device memory*/
	return 0;
}
